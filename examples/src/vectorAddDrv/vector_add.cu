
#include <hip/hip_runtime.h>
/*
 * Name      : VectorAdd
 * Copyright : (c) 2009 Trevor L. McDonell
 * License   : BSD
 *
 * Element-wise addition of two (floating-point) vectors
 */


extern "C"
__global__ void VecAdd(const float *xs, const float *ys, float *out, const unsigned int N)
{
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < N)
        out[idx] = xs[idx] + ys[idx];
}

